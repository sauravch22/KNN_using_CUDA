#include "hip/hip_runtime.h"

//-------------------------------------GPU Implementation of KNN--------------------------------------------------
//---------------------------Train Data store in input.txt and Test data in test.txt------------------------------

#include<iostream>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<stdlib.h>
#include<stdio.h>
#include<thrust/sort.h>
#include<math.h>
#include<hip/hip_runtime.h>
using namespace std;

// Calculating distance in parallel for one test point and all training point 
// Kernal launched with 1*n threads

__global__ void k1(float *gdata,float *gquery,float *gres,int *gid,int N,int count) {
    int id = threadIdx.x;
    //gres[id*2+0] = id;
    gid[id] = id;
    float dist = 0;
    for(int i=1;i<count;i++){
        //printf("%d\t%0.2f\t%0.2f\n",id,gdata[id*count+i],gquery[i]);
        dist += (gdata[id*count+i]-gquery[i])*(gdata[id*count+i]-gquery[i]);
    }
    gres[id] = sqrt(dist);
    //printf("%d %0.2f\n",id,gres[id]);
}
/*__global__ void k(float *data,int N,int count){
        for(int j=0;j<count;j++){
            printf("%d\n",data[threadIdx.x*count+j]);
        }
}*/

//Calculating distances in parallel between all train point and test point .
//kernal launched with m*n threads


__global__ void maxkernal(float *data,float *query,float *dis,int *gid,int N,int count){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int i = id/N;
    int j = id%N;
    //float diss = 0;
    for(int k=1;k<count;k++){
        //printf("%d %0.2f %0.2f %0.2f %0.2f\n",id,data[j*count+k],query[i*count+k],(data[j*count+k]-query[i*count+k]),dis[id]);
        atomicAdd(&dis[id],((data[j*count+k]-query[i*count+k])*(data[j*count+k]-query[i*count+k])));
        //printf("%d %0.2f %0.2f %0.2f %0.2f %0.2f\n",id,data[j*count+k],query[i*count+k],(data[j*count+k]-query[i*count+k]),dis[id],((data[j*count+k]-query[i*count+k])*(data[j*count+k]-query[i*count+k])));
    }
    gid[id] = id;
    dis[id] = sqrt(dis[id]);
}

// Accuracy calculation in parallel

__global__ void Accuracy(int *s1,int *s2,int *counter){
    int id = threadIdx.x;
    //printf("%d %d\n",s1[id],s2[id]);
    int x = 1;
    if(s1[id]==s2[id]){
        atomicAdd(&counter[0],x);
    }
}

// Begin of the main function 


int main(){


    //Reading the train points


    int k=15;
    int N=135;
    int count=0;
    FILE *fp;
    string s[N];
    fp = fopen("input.txt","r");
    char ch = ' ';
    while(ch!='\n'){
        ch = getc(fp);
        if(ch==','){
        count++;
        }
    }
    float *data = (float *)malloc(N*count*sizeof(float));
    for(int i=0;i<N;i++){
        for(int j=0;j<count;j++){
            fscanf(fp,"%f",&data[i*count+j]);
            ch = fgetc(fp);
            //cout<<data[i*count+j]<<"\t";
        }
        char c;
        c = fgetc(fp);
        while(c!='\n'){
            s[i] += c;
            c = fgetc(fp);
        }
        //cout<<s[i]<<"\n";
    }
    fclose(fp);
    float *gdata,*gres,*res;
    int *id,*gid;
    int *fclass;
    /*hipMalloc(&gdata,N*count*sizeof(float));
    hipMemcpy(gdata,data,N*count*sizeof(float),hipMemcpyHostToDevice);
    k<<<1,N>>>(gdata,N,count);*/
    //cout<<"----------------------------------------------------\n";
    
    
    //Reading the test point 


    FILE *op;
    int m=15;
    string s1[m];
    int gsres[m];
    float *query,*gquery;
    float *query2d = (float *)malloc(m*count*sizeof(float));
    fclass = (int *)malloc(m*sizeof(int));
    op = fopen("test.txt","r");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float ms = 0;

    for(int i=0;i<m;i++){
        query = (float *)malloc(count*sizeof(float));
        for(int j=0;j<count;j++){
            fscanf(op,"%f",&query[j]);
            query2d[i*count+j] = query[j];
            ch = fgetc(op);
            //cout<<query[i*count+j]<<"\t";
        }
        char c;
        c = fgetc(op);
        while(c!='\n'){
            s1[i] += c;
            c = fgetc(op);
        }
        if(s1[i]=="Iris-setosa"){
            fclass[i] = 1;
            //cout<<"c1";
        }
        if(s1[i]=="Iris-versicolor"){
            fclass[i] = 2;
            //cout<<"c2";
        }
        if(s1[i]=="Iris-virginica"){
            fclass[i] = 3;
            //cout<<"c3";
        }
        //cout<<s1[i]<<"\n";
        float milliseconds = 0;
        hipEventRecord(start,0);
        hipMalloc(&gquery,count*sizeof(float));
        hipMalloc(&gdata,N*count*sizeof(float));
        hipMalloc(&gres,N*sizeof(float));
        hipMalloc(&gid,N*sizeof(int));
        res = (float *)malloc(N*sizeof(float));
        id = (int *)malloc(N*sizeof(int));
        hipMemcpy(gdata,data,N*count*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(gquery,query,count*sizeof(float),hipMemcpyHostToDevice);

        //Launching one test point to all train point kernal
        
        k1<<<1,N>>>(gdata,gquery,gres,gid,N,count);
        
        hipMemcpy(res,gres,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(id,gid,N*sizeof(int),hipMemcpyDeviceToHost);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        ms += milliseconds;
        thrust::sort_by_key(res, res + N, id);
        int count1,count2,count3;
        count1 = count2 = count3 = 0;

        //voting process of K closest neighbour


        for(int j=0;j<k;j++){
            //cout<<i<<" "<<minKarr[j][0]<<" "<<minKarr[j][1]<<"\n";
            if(s[id[j]]=="Iris-setosa"){
                count1++;
            }
            if(s[id[j]]=="Iris-versicolor"){
                count2++;
            }
            if(s[id[j]]=="Iris-virginica"){
                count3++;
            }
        }
        //cout<<count1<<" "<<count2<<" "<<count3<<"\n";
        if(count1>count2){
            if(count1>count3){
                //count1
                gsres[i] = 1;
            }
            else{
               //count3
                gsres[i] = 3;
            }
        }
        else{
           if(count2>count3){
              //count2
              gsres[i] = 2;
           }
           else{
               //count3
               gsres[i] = 3;
           }
        }
        //cout<<gsres[i]<<"\n";
        //cout<<"---------------------------------------------\n";
    }
    /*for(int i=0;i<m;i++){
        printf("%d\n",fclass[i]);
    }*/
    int *gclass,*ggsres,*gcounter;
    int counter[1];
    counter[0] = 0;
    hipMalloc(&gclass,m*sizeof(int));
    hipMalloc(&ggsres,m*sizeof(int));
    hipMalloc(&gcounter,1*sizeof(int));
    hipMemcpy(gclass,fclass,m*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(ggsres,gsres,m*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gcounter,counter,1*sizeof(int),hipMemcpyHostToDevice);

    // Accuracy calculation 


    Accuracy<<<1,m>>>(gclass,ggsres,gcounter);
    hipMemcpy(counter,gcounter,1*sizeof(int),hipMemcpyDeviceToHost);
    //printf("%d\n",counter[0]);
    float acc = counter[0]*100;
    acc = acc/m;
    
    printf("Basic KNN Time taken in %f millisecond\n",ms);


    //cout<<"Time taken "<<elapsetime<<"\n";
    
    cout<<"Accuracy of KNN "<<acc<<"%"<<"\n";
    
    
    // prediction on random points
    srand(time(0));
    float *points = (float *)malloc(count*sizeof(float));
    for(int j=0;j<count;j++){
        if(j<count-1){
            points[j] = rand()%8;
        }
        else{
            points[j] = rand()%3;
        }
    }
    /*for(int j=0;j<count;j++){
        cout<<points[j]<<"\t";
    }*/
    cout<<"\n";    
    float *dis,*ggdata;
    float *gpoint,*gdis;
    int *gidd;
    int *idd;
    hipMalloc(&gpoint,count*sizeof(float));
    hipMalloc(&ggdata,N*count*sizeof(float));
    hipMalloc(&gdis,N*sizeof(float));
    hipMalloc(&gidd,N*sizeof(int));
    dis = (float *)malloc(N*sizeof(float));
    idd = (int *)malloc(N*sizeof(int));
    hipMemcpy(ggdata,data,N*count*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(gpoint,points,count*sizeof(float),hipMemcpyHostToDevice);

    //Launching one test point to all train point kernal

    k1<<<1,N>>>(gdata,gpoint,gdis,gidd,N,count);
    hipMemcpy(dis,gdis,N*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(idd,gidd,N*sizeof(int),hipMemcpyDeviceToHost);
    thrust::sort_by_key(dis, dis + N, idd);
    int count1,count2,count3; 
    count1 = count2 = count3 = 0;
    
    //voting process of K closest neighbour

    for(int i=0;i<k;i++){
        if(s[idd[i]]=="Iris-setosa"){
            count1++;
        }
        if(s[idd[i]]=="Iris-versicolor"){
            count2++;
        }
        if(s[idd[i]]=="Iris-virginica"){
            count3++;
        }
    }

    //Deciding on voting result

    string prediction;
    if(count1>count2){
        if(count1>count3){
            //count1
            prediction = "Iris-setosa";
        }
        else{
           //count3
           prediction = "Iris-virginica";
        }
   }
   else{
       if(count2>count3){
          //count2
          prediction = "Iris-versicolor";
       }
       else{
           //count3
           prediction = "Iris-virginica";
       }
   }
   cout<<"prediction Result "<<prediction<<"\n";
   

    // More parallelism 
    
    /*for(int i=0;i<m;i++){
        for(int j=0;j<count;j++){
            cout<<query2d[i*count+j]<<"\t";
        }
        cout<<"\n";
    }*/
    
    //One more Knn implementation
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    float milliseconds1 = 0;
    hipEventRecord(start1,0);
    int *id2d,*gid2d;
    int *mres = (int *)malloc(m*sizeof(int));
    float *gquery2d,*gdatam,*gdist,*dist;
    hipMalloc(&gquery2d,m*count*sizeof(float));
    hipMemcpy(gquery2d,query2d,m*count*sizeof(float),hipMemcpyHostToDevice);
    hipMalloc(&gdatam,N*count*sizeof(float));
    hipMemcpy(gdatam,data,N*count*sizeof(float),hipMemcpyHostToDevice);
    dist = (float *)malloc(m*N*sizeof(float));
    hipMalloc(&gdist,m*N*sizeof(float));
    id2d = (int *)malloc(m*N*sizeof(int));
    hipMalloc(&gid2d,m*N*sizeof(int));

    //Distance calculation of KNN through all train and all test points in parallel
    //launching M*N threads
    

    maxkernal<<<m,N>>>(gdatam,gquery2d,gdist,gid2d,N,count);
    hipMemcpy(dist,gdist,m*N*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(id2d,gid2d,m*N*sizeof(int),hipMemcpyDeviceToHost);
    hipEventRecord(stop1,0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&milliseconds1, start1, stop1);
    
    for(int i=0;i<m;i++){
        float *distance = (float *)malloc(N*sizeof(float));
        int *index = (int *)malloc(N*sizeof(int));
        for(int j=0;j<N;j++){
            distance[j] = dist[i*N+j];
            index[j] = id2d[i*N+j];
        }

        //Sorting the K nearest neighbour.

        thrust::sort_by_key(distance, distance + N, index);
        int count1,count2,count3;
        
        //voting for K nearest neighbour

        count1 = count2 = count3 = 0;
        for(int j=0;j<k;j++){
            int p = index[j]%N;
                //cout<<i<<" "<<minKarr[j][0]<<" "<<minKarr[j][1]<<"\n";
                if(s[p]=="Iris-setosa"){
                    count1++;
                }
                if(s[p]=="Iris-versicolor"){
                    count2++;
                }
                if(s[p]=="Iris-virginica"){
                    count3++;
                }
        }
        //cout<<count1<<" "<<count2<<" "<<count3<<"\n";
        if(count1>count2){
            if(count1>count3){
                //count1
                mres[i] = 1;
            }
            else{
               //count3
                mres[i] = 3;
            }
        }
        else{
           if(count2>count3){
              //count2
              mres[i] = 2;
           }
           else{
               //count3
               mres[i] = 3;
           }
        }

        
        //cout<<mres[i]<<"\n";
        //cout<<"\n=========================================================================\n";
        
    
    }

    // Accuracy calculation.
    int *ggclass,*gggsres,*ggcounter;
    int ccounter[1];
    ccounter[0] = 0;
    hipMalloc(&ggclass,m*sizeof(int));
    hipMalloc(&gggsres,m*sizeof(int));
    hipMalloc(&ggcounter,1*sizeof(int));
    hipMemcpy(ggclass,fclass,m*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gggsres,mres,m*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(ggcounter,ccounter,1*sizeof(int),hipMemcpyHostToDevice);
    Accuracy<<<1,m>>>(ggclass,gggsres,ggcounter);
    hipMemcpy(ccounter,ggcounter,1*sizeof(int),hipMemcpyDeviceToHost);
    //printf("%d\n",counter[0]);
    float aacc = ccounter[0]*100;
    aacc = aacc/m;

    
    printf("Time taken %f\n",milliseconds1);

    cout<<"Accuracy of KNN after Max Parallelism "<<acc<<"%"<<"\n";
    
    //cout<<"---------------------------------------------\n";
    
    //Free gpu variables

    hipFree(ggclass);
    hipFree(gggsres);
    hipFree(ggcounter);
    hipFree(gquery2d);
    hipFree(gdatam);
    hipFree(gdis);
    hipFree(gdist);
    hipFree(gid);
    hipFree(gid2d);
    hipFree(gpoint);
    hipFree(gquery);
    hipFree(gdata);
    hipFree(gcounter);
    hipFree(gclass);
    hipFree(gsres);
    hipFree(gres);
    hipFree(gidd);
    hipFree(ggdata);

    //Free Cpu variables

    free(data);
    free(fclass);
    free(res);
    free(id);
    free(query);
    free(query2d);
    free(points);
    free(idd);
    free(dis);
    free(id2d);
    free(mres);
    free(dist);
    

    //---------------------------++++++++++++++++++++++++----------------------------
    hipDeviceSynchronize();
    return 0;
}
