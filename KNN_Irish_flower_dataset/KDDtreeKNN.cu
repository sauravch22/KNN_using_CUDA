#include "hip/hip_runtime.h"

#include<iostream>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<algorithm>
#include<thrust/sort.h>
#include<math.h>
#include<stdio.h>
using namespace std;
struct tree{
    int id;
    int leftid;
    int parent;
    float filter;
    int rightid;
    int pos;
    int startpos;
    int endpos;
}Maptree[30];
__global__ void distance(float *data,float *query,float *dis,int *id,int count,int start){
    int idt = threadIdx.x;
    idt = threadIdx.x+start;
    //printf("%d\n",idt);
    float dist = 0;
    for(int i=1;i<count;i++){
        dist += (data[idt*count+i]-query[i])*(data[idt*count+i]-query[i]);
    }
    dist = sqrt(dist);
    id[threadIdx.x] = data[idt*count+0];
    dis[threadIdx.x] = dist ;
}
__global__ void Accuracy(int *s1,int *s2,int *counter){
    int id = threadIdx.x;
    //printf("%d %d\n",s1[id],s2[id]);
    int x = 1;
    if(s1[id]==s2[id]){
        atomicAdd(&counter[0],x);
    }
}
void KDDpartition(float *index,float *data,int points,int count,int front,int N,int time){
    //cout<<"\n========================================================================\n";
    Maptree[time].id = time;
    int Noofitems = Maptree[time].endpos - Maptree[time].startpos;
    //cout<<Noofitems<<"\n";
    if(Noofitems<points){
        return ;
    }
    float **decide = (float **)malloc(count*sizeof(float*));
    float *mean = (float *)malloc(count*sizeof(float));
    float *var = (float *)malloc(count*sizeof(float));
    for(int i=0;i<count;i++){
        decide[i] = (float *)malloc(N*sizeof(float));
        for(int j=front;j<N;j++){
            decide[i][j] = data[j*count+i];
            mean[i] += decide[i][j]; 
        }
        mean[i] = mean[i]/N;
    }
    for(int i=0;i<count;i++){
        for(int j=front;j<N;j++){
            var[i] +=(decide[i][j]-mean[i])*(decide[i][j]-mean[i]);
        }
        var[i] = var[i]/N;
    }
    float Max = 0;
    int pos = 0;
    for(int i=1;i<count;i++){
        if(Max<var[i]){
            Max = var[i];
            pos = i;
        }
    }
    //cout<<Max<<" "<<pos<<"\n";
    float *cdata = (float *)malloc(N*count*sizeof(float));
    sort(decide[pos]+front,decide[pos]+N);
    for(int i=front;i<N;i++){
        //cout<<decide[pos][i]<<"\t";
    }
    //cout<<"\n";
    int mid = (N-front)/2;
    float Median = decide[pos][front+mid];
    //cout<<mid<<" "<<Median<<"\n";
    int start,last;
    start = Maptree[time].startpos;
    last = Maptree[time].endpos;
    Maptree[time].filter = Median;
    Maptree[time].pos = pos;
    for(int i=front;i<N;i++){
        if(data[i*count+pos]<Median){
            for(int j=0;j<count;j++){
                cdata[start*count+j] = data[i*count+j];
            }
            start++;
        }
        else{
            for(int j=0;j<count;j++){
                cdata[last*count+j] = data[i*count+j];
            }
            last--;
        }
    }
    //cout<<start<<" "<<last<<"\n";
    /*for(int i=front;i<N;i++){
        cout<<i<<"\t";
        for(int j=0;j<count;j++){
            cout<<cdata[i*count+j]<<"\t";
        }
        cout<<"\n";
    }*/
    int left = 2*time;
    int right = 2*time+1;
    Maptree[time].leftid = left;
    Maptree[time].rightid = right;
    Maptree[left].parent = time;
    Maptree[right].parent = time;
    Maptree[left].startpos = front;
    Maptree[left].endpos = last;
    Maptree[right].startpos = start;
    Maptree[right].endpos = Maptree[time].endpos;
    //cout<<Maptree[left].startpos<<" "<<Maptree[left].endpos<<" "<<Maptree[right].startpos<<" "<<Maptree[right].endpos<<"\n";
    for(int i=front;i<N;i++){
        //cout<<i<<"\t";
        for(int j=0;j<count;j++){
            data[i*count+j] = cdata[i*count+j];
            //cout<<data[i*count+j]<<"\t";
        }
        //cout<<"\n";
    }
    KDDpartition(index,data,points,count,Maptree[left].startpos,last+1,left);
    KDDpartition(index,data,points,count,Maptree[right].startpos,Maptree[right].endpos+1,right);
}
void search(float *data,float *query,int points,int count,int N,int m,int time,int k,string s[],string s1[]){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms = 0;
    
    int noofelements = Maptree[time].endpos - Maptree[time].startpos;
    //cout<<noofelements<<"\n";
    int x = time;
    int *fclass = (int *)malloc(m*sizeof(int));
    int *res = (int *)malloc(m*sizeof(int)); 
    float *line = (float *)malloc(count*sizeof(float));
    for(int i=0;i<m;i++){
        if(s1[i]=="Iris-setosa"){
            fclass[i] = 1;
            //cout<<"c1";
        }
        if(s1[i]=="Iris-versicolor"){
            fclass[i] = 2;
            //cout<<"c2";
        }
        if(s1[i]=="Iris-virginica"){
            fclass[i] = 3;
            //cout<<"c3";
        }
        for(int j=0;j<count;j++){
            line[j] = query[i*count+j];
            //cout<<line[j]<<"\t"; 
        }
        //cout<<"\n";   
        while(noofelements>points){
            int dim = Maptree[x].pos;
            float Median = Maptree[x].filter;
            if(query[i*count+dim]<Median){
                x = Maptree[x].leftid;
            }
            else{
                x = Maptree[x].rightid;
            }
            noofelements = Maptree[x].endpos - Maptree[x].startpos;
        }
        x = Maptree[x].parent;
        int st = Maptree[x].startpos;
        int et = Maptree[x].endpos;
        //cout<<x<<" "<<st<<" "<<et<<"\n";
        float *gdata,*gquery,*dis,*gdis;
        int *id,*gid;
        id = (int *)malloc(N*sizeof(int));
        dis = (float *)malloc(N*sizeof(float));
        float milliseconds = 0;
        hipEventRecord(start,0);
        hipMalloc(&gid,N*sizeof(int));
        hipMalloc(&gdis,N*sizeof(float));
        hipMalloc(&gdata,N*count*sizeof(float));
        hipMalloc(&gquery,count*sizeof(float));
        hipMemcpy(gdata,data,N*count*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(gquery,line,count*sizeof(float),hipMemcpyHostToDevice);
        //cout<<"\n------------------\n";
        distance<<<1,(et-st)>>>(gdata,gquery,gdis,gid,count,st);
        hipMemcpy(dis,gdis,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(id,gid,N*sizeof(int),hipMemcpyDeviceToHost);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        ms += milliseconds;
        thrust::sort_by_key(dis, dis + (et-st), id);
        int count1,count2,count3;
        count1 = count2 = count3 = 0;
        for(int j=0;j<k;j++){
            //cout<<id[j]<<" "<<dis[j]<<"\n";
            if(id[j]<=50 && id[j]>0){
                count1++;
            }
            if(id[j]>50 && id[j]<=100){
                count2++;
            }
            if(id[j]<=150 && id[j]>100){
                count3++;
            }
        }
        //cout<<"------------------"<<count1<<" "<<count2<<" "<<count3<<"\n";
        if(count1>count2){
            if(count1>count3){
                //count1
                res[i] = 1;
            }
            else{
               //count3
               res[i] = 3;
            }
        }
        else{
           if(count2>count3){
              //count2
               res[i] = 2;
           }
           else{
               //count3
               res[i] = 3;
           }
        }
        x = time;
        noofelements = Maptree[x].endpos - Maptree[x].startpos; 
    }
    int *gclass,*ggsres,*gcounter;
    int counter[1];
    counter[0] = 0;
    hipMalloc(&gclass,m*sizeof(int));
    hipMalloc(&ggsres,m*sizeof(int));
    hipMalloc(&gcounter,1*sizeof(int));
    hipMemcpy(gclass,fclass,m*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(ggsres,res,m*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gcounter,counter,1*sizeof(int),hipMemcpyHostToDevice);
    Accuracy<<<1,m>>>(gclass,ggsres,gcounter);
    hipMemcpy(counter,gcounter,1*sizeof(int),hipMemcpyDeviceToHost);
    float acc = counter[0]*100;
    acc = acc/m;
    printf("Total Execution time %f in millisecond\n",ms);
    cout<<"Accuracy of KD tree implementation of KNN "<<acc<<"% \n";

}



////////////////////////
void searchprediction(float *data,float *query,int points,int count,int N,int m,int time,int k,string s[],string s1[]){
    int noofelements = Maptree[time].endpos - Maptree[time].startpos;
    //cout<<noofelements<<"\n";
    int x = time;
    string sf = "";
    while(noofelements>points){
            int dim = Maptree[x].pos;
            float Median = Maptree[x].filter;
            if(query[dim]<Median){
                x = Maptree[x].leftid;
            }
            else{
                x = Maptree[x].rightid;
            }
            noofelements = Maptree[x].endpos - Maptree[x].startpos;
        }
        x = Maptree[x].parent;
        int st = Maptree[x].startpos;
        int et = Maptree[x].endpos;
        //cout<<x<<" "<<st<<" "<<et<<"\n";
        float *gdata,*gquery,*dis,*gdis;
        int *id,*gid;
        id = (int *)malloc(N*sizeof(int));
        dis = (float *)malloc(N*sizeof(float));
        hipMalloc(&gid,N*sizeof(int));
        hipMalloc(&gdis,N*sizeof(float));
        hipMalloc(&gdata,N*count*sizeof(float));
        hipMalloc(&gquery,count*sizeof(float));
        hipMemcpy(gdata,data,N*count*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(gquery,query,count*sizeof(float),hipMemcpyHostToDevice);
        //cout<<"\n------------------\n";
        distance<<<1,(et-st)>>>(gdata,gquery,gdis,gid,count,st);
        hipMemcpy(dis,gdis,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(id,gid,N*sizeof(int),hipMemcpyDeviceToHost);
        thrust::sort_by_key(dis, dis + (et-st), id);
        int count1,count2,count3;
        count1 = count2 = count3 = 0;
        for(int j=0;j<k;j++){
            //cout<<id[j]<<" "<<dis[j]<<"\n";
            if(id[j]<=50 && id[j]>0){
                count1++;
            }
            if(id[j]>50 && id[j]<=100){
                count2++;
            }
            if(id[j]<=150 && id[j]>100){
                count3++;
            }
        }
        //cout<<"------------------"<<count1<<" "<<count2<<" "<<count3<<"\n";
        if(count1>count2){
            if(count1>count3){
                //count1
                sf = "Iris-setosa";
            }
            else{
               //count3
               sf = "Iris-virginica";
            }
        }
        else{
           if(count2>count3){
              //count2
               sf = "Iris-versicolor";
           }
           else{
               //count3
               sf = "Iris-virginica";
           }
        } 
    cout<<"Predicted output for random point"<<sf<<"\n";
}

////////////////////////





int main(){
    int points = 20;
    int k = 15;
    cout<<"KDD Tree implementation of KNN Algorithm\n";
    FILE *fp;
    int N = 135;
    int count = 0 ; 
    fp = fopen("input.txt","r");
    char ch = ' ';
    while(ch!='\n'){
        ch = getc(fp);
        if(ch==','){
        count++;
        }
    }
    string s[N];
    float *data = (float *)malloc(N*count*sizeof(float));
    for(int i=0;i<N;i++){
        for(int j=0;j<count;j++){
            fscanf(fp,"%f",&data[i*count+j]);
            ch = fgetc(fp);
            //cout<<data[i*count+j]<<"\t";
        }
        char c;
        c = fgetc(fp);
        while(c!='\n'){
            s[i]+=c;
            c = fgetc(fp);
        }
        //cout<<s[i]<<"\n";
    }
    //cout<<"\n=================================================\n";
    int m =15;
    float *query = (float *)malloc(m*count*sizeof(float));
    FILE *op;
    string s1[m];
    op = fopen("test.txt","r");
    for(int i=0;i<m;i++){
        for(int j=0;j<count;j++){
            fscanf(op,"%f",&query[i*count+j]);
            ch = fgetc(op);
            //cout<<query[i*count+j]<<"\t";
        }
        char c;
        c = fgetc(op);
        while(c!='\n'){
            s1[i] += c;
            c = fgetc(op);
        }
        //cout<<s1[i]<<"\n";
    }
    float *index = (float *)malloc(N*2*sizeof(float));
    //Grouping all data
    for(int i=0;i<N;i++){
       index[i*2+0] = 1;
       index[i*2+1] = data[i*count+0];
       //cout<<index[i*2+0]<<" "<<index[i*2+1]<<"\n";
    }
    Maptree[1].id = 1;
    Maptree[1].leftid = 0;
    Maptree[1].filter = 0;
    Maptree[1].rightid = 0;
    Maptree[1].pos = 0;
    Maptree[1].parent = -1;
    Maptree[1].startpos = 0;
    Maptree[1].endpos = 134;
    
    KDDpartition(index,data,points,count,0,N,1);
    //cout<<"\n==============================================================\n";
    /*for(int i=0;i<N;i++){
        for(int j=0;j<count;j++){
           // cout<<data[i*count+j]<<"\t";
        }
        //cout<<"\n";
    }*/
    search(data,query,points,count,N,m,1,k,s,s1);
    
    

    srand(time(0));
    float *point = (float *)malloc(count*sizeof(float));
    for(int j=0;j<count;j++){
        if(j<count-1){
            point[j] = rand()%8;
        }
        else{
            point[j] = rand()%3;
        }
        //cout<<point[j]<<"\t";
    }
    
    searchprediction(data,point,points,count,N,m,1,k,s,s1);
    hipDeviceSynchronize();
    return 0;
}