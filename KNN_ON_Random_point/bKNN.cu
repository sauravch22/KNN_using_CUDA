#include "hip/hip_runtime.h"

#include<iostream>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<stdlib.h>
#include<stdio.h>
#include<thrust/sort.h>
#include<math.h>
#include<hip/hip_runtime.h>
using namespace std;
__global__ void k1(long *gdata,long *gquery,long *gres,int *gid,int N,int count) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;;
    //gres[id*2+0] = id;
    gid[id] = id;
    float dist = 0;
    for(int i=1;i<count-1;i++){
        //printf("%d\t%0.2f\t%0.2f\n",id,gdata[id*count+i],gquery[i]);
        dist += (gdata[id*count+i]-gquery[i])*(gdata[id*count+i]-gquery[i]);
    }
    gres[id] = sqrt(dist);
    //printf("%d %0.2f\n",id,gres[id]);
}
__global__ void maxk(long *data,long *query,long *res,int *gid,int N,int count){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int i = id%N;
    int j = id/N;
    float dis = 0;
    for(int k=1;k<count-1;k++){
        dis +=((data[i*count+k]-query[j*count+k])*(data[i*count+k]-query[j*count+k]));
    }
    //printf("%d\n",id);
    res[id] = sqrt(dis);
    gid[id] = id;
}
__global__ void Accuracy(long *query,long *result,int count,int *counter){
    int id = threadIdx.x;
    //printf("%d %d\n",s1[id],s2[id]);
    int x = 1;
    if(query[id*count+10]==result[id]){
        atomicAdd(&counter[0],x);
    }
}

int main(){
    int k = 3 ;
    FILE *fp;
    int N = 10000;
    int count = 11;
    fp = fopen("binput.txt","r");
    char ch = ' ';
    long *data = (long *)malloc(N*count*sizeof(long));
    for(int i=0;i<N;i++){
        for(int j=0;j<count;j++){
            fscanf(fp,"%ld",&data[i*count+j]);
            ch = fgetc(fp);
            //cout<<data[i*count+j]<<"\t";
        }
        //cout<<"\n";
    }
    int m = 100;
    count = 11;
    FILE *op;
    op = fopen("bitest.txt","r");
    long *query = (long *)malloc(m*count*sizeof(long));
    for(int i=0;i<m;i++){
        for(int j=0;j<count;j++){
            fscanf(op,"%ld",&query[i*count+j]);
            ch = fgetc(op);
            //cout<<query[i*count+j]<<"\t";
        }
        //cout<<"\n";
    }
    long *result = (long *)malloc(m*sizeof(long));
    long *gquery,*gdata,*res,*gres;
    int *id,*gid;
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    float ms = 0;
    for(int i=0;i<m;i++){
        long *point = (long *)malloc(count*sizeof(long));
        for(int j=0;j<count;j++){
            point[j] = query[i*count+j];
        }
        float milliseconds1 = 0;
        hipEventRecord(start1,0);
        hipMalloc(&gquery,count*sizeof(long));
        hipMalloc(&gdata,N*count*sizeof(long));
        hipMalloc(&gres,N*sizeof(long));
        hipMalloc(&gid,N*sizeof(int));
        res = (long *)malloc(N*sizeof(long));
        id = (int *)malloc(N*sizeof(int));
        hipMemcpy(gdata,data,N*count*sizeof(long),hipMemcpyHostToDevice);
        hipMemcpy(gquery,point,count*sizeof(long),hipMemcpyHostToDevice);

        //Launching one test point to all train point kernal


        k1<<<16,N/16>>>(gdata,gquery,gres,gid,N,count);
        hipMemcpy(res,gres,N*sizeof(long),hipMemcpyDeviceToHost);
        hipMemcpy(id,gid,N*sizeof(int),hipMemcpyDeviceToHost);
        hipEventRecord(stop1,0);
        hipEventSynchronize(stop1);
        hipEventElapsedTime(&milliseconds1, start1, stop1);
        ms+=milliseconds1;
        thrust::sort_by_key(res, res + N, id);
        //cout<<"\n============================\n";
        int count1,count2;
        count1 = count2 = 0;
        for(int j=0;j<k;j++){
            //cout<<i<<" "<<id[j]<<" "<<res[j]<<"\n";
            //cout<<id[j]<<" "<<data[id[j]*count+10]<<"\n";
            if(data[id[j]*count+10]==2){
                count1++;
            }
            if(data[id[j]*count+10]==4){
                count2++;
            }
            
        }
        //cout<<count1<<" "<<count2<<"\n";
        if(count1>count2){
            result[i] = 2;
        }
        else{
            result[i] = 4;
        }
    }
    int *gcounter;
    int counter[1];
    long *gresult,*ggquery;
    hipMalloc(&gresult,m*sizeof(long));
    hipMalloc(&ggquery,m*count*sizeof(long));
    counter[0] = 0;
    hipMalloc(&gcounter,1*sizeof(int));

    hipMemcpy(gcounter,counter,1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(ggquery,query,m*count*sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(gresult,result,m*sizeof(long),hipMemcpyHostToDevice);

    Accuracy<<<1,m>>>(ggquery,gresult,count,gcounter);
    hipMemcpy(counter,gcounter,1*sizeof(int),hipMemcpyDeviceToHost);
    
    printf(" Total time taken %f\n",ms);
    //cout<<counter[0];
    float acc = counter[0]*100;
    acc = acc/m;
    cout<<"Accuracy of KNN "<<acc<<"\n";
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    
    int *id2d,*gid2d;
    long *gdata2d,*gquery2d,*gres2d,*res2d;
    hipMalloc(&gid2d,m*N*sizeof(int));
    id2d = (int *)malloc(m*N*sizeof(int));
    res2d = (long *)malloc(m*N*sizeof(long));
    hipMalloc(&gres2d,m*N*sizeof(long));
    hipMalloc(&gdata2d,N*count*sizeof(long));
    hipMalloc(&gquery2d,m*count*sizeof(long));
    hipMemcpy(gdata2d,data,N*count*sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(gquery2d,query,m*count*sizeof(long),hipMemcpyHostToDevice);
    
    maxk<<<16*m,N/16>>>(gdata2d,gquery2d,gres2d,gid2d,N,count);
    
    hipMemcpy(id2d,gid2d,m*N*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(res2d,gres2d,m*N*sizeof(long),hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Total time taken %f\n",milliseconds);
    for(int i=0;i<m;i++){
        //cout<<"Line"<<i<<"\t";
        long *dist = (long *)malloc(N*sizeof(long));
        int *im = (int *)malloc(N*sizeof(int));
        for(int j=0;j<N;j++){
            //cout<<res2d[i*N+j]<<"\t";
            im[j] = id2d[i*N+j]%N;
            dist[j] = res2d[i*N+j];
        }
        thrust::sort_by_key(dist, dist + N, im);
        int count1,count2;
        count1 = count2 = 0;
        for(int j=0;j<k;j++){
            //cout<<im[j]<<"\t";
            if(data[im[j]*count+10]==2){
                count1++;
            }
            if(data[im[j]*count+10]==4){
                count2++;
            }
        }
        if(count1>count2){
            result[i] = 2;
        }
        else{
            result[i] = 4;
        }
        //cout<<result[i]<<"\n";
        //cout<<count1<<" "<<count2<<"\n";
    }

    
    int *ggcounter;
    int ccounter[1];
    long *ggresult,*gggquery;
    hipMalloc(&ggresult,m*sizeof(long));
    hipMalloc(&gggquery,m*count*sizeof(long));
    ccounter[0] = 0;
    hipMalloc(&ggcounter,1*sizeof(int));

    hipMemcpy(ggcounter,ccounter,1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gggquery,query,m*count*sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(ggresult,result,m*sizeof(long),hipMemcpyHostToDevice);

    Accuracy<<<1,m>>>(gggquery,ggresult,count,ggcounter);
    hipMemcpy(ccounter,ggcounter,1*sizeof(int),hipMemcpyDeviceToHost);
    
    float acc1 = ccounter[0]*100;
    acc1 = acc1/m;

    cout<<"Accuracy of KNN "<<acc1<<"\n";

    hipDeviceSynchronize();
    return 0;
}